#include "hip/hip_runtime.h"
#pragma once
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <iostream>
#include <thrust/device_ptr.h>
#include <cmath>
#include <assert.h>
#include <hip/hip_runtime.h>
#include ""
#include "random"
 
// generateRays in CPU 
// copy rays_direction into Struct Ray
// copy num_rays Ray struct into GPU

// need to figure it out how create bunch of rays in main function

// Done
float** generateRays(int num_rays) {
	// all_rays_directions store num_rays by 3 matrix
	float** all_rays_directions;
	all_rays_directions = new float*[num_rays];
		
	for (int i = 0; i < num_rays; i++)
	{
		all_rays_directions[i] = new float [3];
	}

	std::default_random_engine generator;
	std::uniform_real_distribution<float> distribution(-1.0, 1.0);
	std::uniform_real_distribution<float> dist(0.0, 1.0);
	const long double PI = 3.14159265358979323846264338328;
	float* phi;
	phi = new float[num_rays];
	for (int i = 0; i < num_rays; i++)
	{
		all_rays_directions[i][0] = distribution(generator);
		//std::cout << all_rays_directions[i][0] << std::endl;
		phi[i] = dist(generator);
		phi[i] = 2 * PI * phi[i];

	}
	

	for (int i = 0; i < num_rays; i++) {

		double coeff = sqrt(1 - all_rays_directions[i][0]*all_rays_directions[i][0]);
		
			all_rays_directions[i][1] = coeff*sin(phi[i]);
			all_rays_directions[i][2] = coeff*cos(phi[i]);
		

	}

	return all_rays_directions;


}

struct Ray
{
	int ray_sign;
	float ray_energy;
	float ray_distance;
	float ray_direction[3];

	float ray_start_point[3];
	Ray(float* start_point, float* direction) {
		ray_energy = 1;
		ray_sign = 1;
		ray_distance = 0;
		for (int i = 0; i < 3; i++)
		{
			ray_direction[i] = direction[i];
			ray_start_point[i] = start_point[i];

		}
		
		for (int i = 0; i < 3; i++)
		{
			std::cout << "initialized ray_direction: " << ray_direction[i] << std::endl;
			//std::cout << "initialized start_point: " << ray_start_point[i] << std::endl;

		}
		//ray_direction = direction;
	}
};

// allocate one room info in CPU and keep one copy in GPU
// how to initialize surfaces and normal_surfaces
struct Room {
	float room_dimension[3];
	float listener_radius;
	float source[3];
	float l_position[3];
	float surfaces[6][4];
	float normal_surfaces[6][3];

	Room(float*source, float* lis_position, float* room_dim) {
		listener_radius = 0.2;
		//printf("radius in constructor: %f", listener_radius);
		for (int i = 0; i < 3; i++)
		{
			room_dimension[i] = room_dim[i];
			source[i] = source[i];
			l_position[i] = lis_position[i];
		}
		/*for (int idx = 0; idx < 6; idx++)
		{
			for (int col = 0; col < 4; col++)
			{
				if (col < 3)
				{
					normal_surfaces[idx][col] = normal_surfaces[idx][col];
					surfaces[idx][col] = surfaces[idx][col];
					
				}
				surfaces[idx][col] = surfaces[idx][col];
				printf("this is surfaces matrix:%f\n", surfaces[idx][col]);
			}
		}*/

		/*for (int i = 0; i < 3; i++)
		{
			std::cout << "this is init room_dim: " << room_dimension[i] << std::endl;
			std::cout << "this is init srouce: " << source[i] << std::endl;
			std::cout << "this is init l_position: " << l_position[i] << std::endl;
		}
    */


	}
};
// helper function
// calDist checked
__device__ float  calPointsDistance(float* point1, float* point2) {
	float distance;
	for (int i = 0; i < 3; i++)
	{
		float d = point1[i] - point2[i];
		/*printf("this is the point1 in cal function: %f\n", point1[i]);
		printf("this is the point2 in cal function: %f\n", point2[i]);
		printf("this is the dist between two points: %f\n", distance);*/
		distance = distance + d*d;

	}
	distance = sqrt(distance);
	return distance;

}  // checked
// cal dot product between vectors
__device__ float calDotProduct(float* vec1, float* vec2) {
	float dot_product = 0;
	// here calculate the dot product between 
	for (int i = 0; i < 3; i++)
	{
		dot_product = dot_product + vec1[i] * vec2[i];
	}

	return dot_product;

}

// Ray related function  // done
__device__ void setStartingPoint(Ray* dev_ray, float* new_start_point) {

	for (int i = 0; i < 3; i++)
	{
		
		dev_ray->ray_start_point[i] = new_start_point[i];
		//printf("new point in set function: %f\n", dev_ray->ray_start_point[i]);

	}
}
__device__ float* getStartingPoint(Ray* dev_ray) {
	//float point[3];
	//for (int i = 0; i < 3; i++)
	//{
	//	//printf("starting point in get function: %f\n", dev_ray->ray_start_point[i]);
	//	point[i] = dev_ray->ray_start_point[i];
	//}
	/*for (int i = 0; i < 3; i++)
	{
		printf("starting point in get function: %f\n", dev_ray->ray_start_point[i]);
		
	}*/
	return dev_ray->ray_start_point;
	//return point;
}
__device__ void setSign(Ray* dev_ray, int sign) {
	dev_ray->ray_sign = sign;
}
__device__ int getSign(Ray* dev_ray) {
	return dev_ray->ray_sign;
}
__device__ void setEnergy(Ray* dev_ray, float energy) {
	dev_ray->ray_energy = energy;
}
__device__ void setDistance(Ray* dev_ray, float dist) {
	dev_ray->ray_distance = dist;
}
__device__ float getDistance(Ray* dev_ray) {
	return dev_ray->ray_distance;
}
__device__ float getEnergy(Ray* dev_ray) {
	return dev_ray->ray_energy;
}
__device__ float* getRayDirection(Ray* dev_ray) {
	return dev_ray->ray_direction;
}
__device__ void setRayDirection(Ray* dev_ray, float* new_direction) {
	for (int i = 0; i < 3; i++)
	{

		dev_ray->ray_direction[i] = new_direction[i];

	}
}

// Room related functoin
//done
__device__ float* getListPos(Room* d_room) {
	/*for (int i = 0; i < 3; i++)
	{
		printf("lis point in get function: %f\n", d_room->l_position[i]);
	}*/
	return d_room->l_position;
}
__device__ bool isPointOnEdge(Room*d_room, float* point) {

	float room_dim_x = d_room->room_dimension[0];
	float room_dim_y = d_room->room_dimension[1];
	float room_dim_z = d_room->room_dimension[2];
	if (point[0] == 0 && (point[2] == 0 || point[2] == room_dim_z)) {
		return true;
	}

	if (point[0] == room_dim_x && (point[2] == 0 || point[2] == room_dim_z)) {
		return true;
	}

	if (point[0] == 0 && (point[1] == 0 || point[1] == room_dim_y)) {
		return true;
	}

	if (point[0] == room_dim_x && (point[1] == 0 || point[1] == room_dim_y)) {
		return true;
	}

	if (point[1] == 0 && (point[2] == 0 || point[2] == room_dim_z)) {
		return true;
	}

	if (point[1] == room_dim_y && (point[2] == 0 || point[2] == room_dim_z)) {
		return true;
	}

	return false;
}
__device__ float* getRoomDimension(Room* d_room) {
	return d_room->room_dimension;
}

// isInSphere checked
__device__ bool isInSphere(Ray* dev_ray, Room* d_room) {

	
	float *start_point = getStartingPoint(dev_ray);
	for (int i = 0; i < 3; i++)
	{
		printf("start_point in isInSphere function: %f\n", start_point[i]);
	}
	float *lis_point = getListPos(d_room);
	
	float point_listen_vec[3];
	float* direction = getRayDirection(dev_ray);
	float norm_point_listen_vec = calPointsDistance(start_point, lis_point);
	float dot_product = 0;
	// here calculate the dot product between point_listen_vec and direction
	for (int i = 0; i < 3; i++)
	{   
		//printf("this is start_point: %f\n", start_point[i]);
		//printf("this is lis_point in bool function: %f\n", lis_point[i]);
		point_listen_vec[i] = start_point[i] - lis_point[i];
		//printf("this is lis_point in point_lis_vec: %f\n", point_listen_vec[i]);
		dot_product = dot_product + point_listen_vec[i] * direction[i];
	}
	/*for (int j = 0; j < 3; j++)
	{
		printf("this is start_point: %f\n", start_point[j]);
		printf("this is direction: %f\n", direction[j]);
		printf("this is point_lis_vec: %f\n", point_listen_vec[j]);
	}*/
	
	float b = 2 * dot_product;
	float a = 1; 
	float c = norm_point_listen_vec*norm_point_listen_vec - d_room->listener_radius * d_room->listener_radius;
	//printf("this is c: %f\n", c);
	float d = b * b - 4 * a * c;
	printf("this is d: %f\n", d);
	if (d < 0) {
		return false;
	}

	return true;
}
// checked with one example
__device__ int isValidIntersectionPoint(Room* d_room, float* point, int id) {


	float* room_dim = getRoomDimension(d_room);
	float room_dim_x = room_dim[0];
	float room_dim_y = room_dim[1];
	float room_dim_z = room_dim[2];
	const float TOLERANCE = 0.00001;

	float x_diff = point[0] - room_dim_x;
	float y_diff = point[1] - room_dim_y;
	float z_diff = point[2] - room_dim_z;
	/*printf("this is the x_diff: %f\n", x_diff);
	printf("this is the y_diff: %f\n", y_diff);
	printf("this is the z_diff: %f\n", z_diff);*/

	if (fabs(x_diff) <= TOLERANCE) {

		x_diff = 0;
		point[0] = room_dim_x;

	}

	if (fabs(point[0]) <= TOLERANCE) {
		point[0] = 0;
	}

	if (fabs(y_diff) <= TOLERANCE) {
		y_diff = 0;
		point[1] = room_dim_y;
	}

	if (fabs(point[1]) <= TOLERANCE) {
		point[1] = 0;
	}

	if (fabs(z_diff) <= TOLERANCE) {
		z_diff = 0;
		point[2] = room_dim_z;

	}

	if (fabs(point[2]) <= TOLERANCE) {
		point[2] = 0;
	}

	if (x_diff <= 0 && y_diff <= 0 && z_diff <= 0 && point[0] >= -TOLERANCE && point[1] >= -TOLERANCE && point[2] >= -TOLERANCE) {
		printf("this is the id inside isValidfunction:%d\n", id);
		return id;

	}

	return -1;
}
// checked with one example
__device__ int getCurrentValudSurfaceID(Ray* dev_ray, float* point, Room* d_room) {
	
	float* room_dim = getRoomDimension(d_room);
	
	float surfaces[6][4] = {
		{ 0, 0, 1, 0 },
		{ 0, 0, 1, -room_dim[2] },
		{ 1, 0, 0, 0 },
		{ 1, 0, 0, -room_dim[0] },
		{ 0, 1, 0, 0 },
		{ 0, 1, 0, -room_dim[1] }
	};
	float normal_surfaces[6][3] = { { 0, 0, 1 }, { 0, 0, -1 }, { 1, 0, 0 }, { -1, 0, 0 }, { 0, 1, 0 }, { 0, -1, 0 } };
	float* direction_vector = getRayDirection(dev_ray);
	float intersection_point[3];
	int surface_id = 0;
	/*for (int i = 0; i < 3; i++)
	{
		printf("this is the room_dim: %f\n", room_dim[i]);
		printf("this is the point: %f\n", point[i]);
		printf("this is the direction: %f\n", direction_vector[i]);
	}*/
	for ( surface_id = 0; surface_id < 6; surface_id++) {

		float dot_product = calDotProduct(direction_vector, normal_surfaces[surface_id]);
		printf("this is the all surface_id: %d\n", surface_id);
		
		//printf("this is the dot_product: %f\n", dot_product);
		if (dot_product < 0) {

				float* surface;
				surface = surfaces[surface_id];
				
				float numerator = 0;
				float denominator = 0;
				for (int i = 0; i < 4; i++)
				{
					if (i < 3) {
						denominator = denominator + surface[i] * direction_vector[i];
						numerator = numerator + surface[i] * point[i];
					}
					else {
						numerator = numerator + surface[i];
					}

				}

				/*printf("this is numerator: %f\n", numerator);
				printf("this is denominator: %f\n", denominator);*/

				for (int k = 0; k < 3; k++)
				{
					//printf("KKKK: %d\n", k);
					intersection_point[k] = point[k] - (numerator / denominator) * direction_vector[k];
					/*printf("this is the point: %f\n", point[k]);
					printf("this is the direction: %f\n", direction_vector[k]);*/
					
					//printf("this is intersection point: %f\n", intersection_point[k]);
				}

				
				//printf("this is the didid surface_id: %d\n", surface_id);
				//printf("this is the dadada surface_id: %d\n", surface_id);
				int valid;
				valid = isValidIntersectionPoint(d_room, intersection_point, surface_id);
				
				/*if (isValidIntersectionPoint(d_room, intersection_point, surface_id) && point != intersection_point) {
					setStartingPoint(dev_ray, intersection_point);
					printf("this is the valid surface_id: %f\n", surface_id);
					for (int k = 0; k < 3; k++)
					{
						printf("this is intersection point: %f\n", intersection_point[k]);
						
					}
					
					return surface_id;
				}*/
				
				if (valid != -1 && point != intersection_point)
				{
					//printf("VALID: %d\n", valid);
					/*for (int k = 0; k < 3; k++)
					{
						printf("this is the intersection point: %f\n", intersection_point[k]);

					}*/
					setStartingPoint(dev_ray, intersection_point);
					
					/*for (int k = 0; k < 3; k++)
					{
						printf("this is the point inside loop: %f\n", point[k]);

					}*/
					return valid;
				}

				if (valid == -1)
				{
					printf("is no valid intersection point detected in isValidIntersectionPoint funtion");
				}

				if (point == intersection_point)
				{
					printf("something worong");
				}

		}
		
	}
	return -1;
}
// checked
__device__ float* calReflecRay(float* ray_direction, int surface_id) {
	float reflected_ray[3];
	float normal_surfaces[6][3] = { { 0, 0, 1 }, { 0, 0, -1 }, { 1, 0, 0 }, { -1, 0, 0 }, { 0, 1, 0 }, { 0, -1, 0 } };
	float temp[3];
	
	for (int i = 0; i < 3; i++)
	{
		//printf("this is the normal_surface: %f\n", normal_surfaces[surface_id][i]);
		temp[i] = -ray_direction[i] * normal_surfaces[surface_id][i]*2;
		//printf("this is the temp: %f\n", temp[i]);
		reflected_ray[i] = temp[i] * normal_surfaces[surface_id][i] + ray_direction[i];

	}

	return reflected_ray;
}
__device__ float absorbEnergy(float energy, int surface_id) {
	float decay_coeff;
	float left_energy;
	if (surface_id == 0) {
		decay_coeff = 0.17;
	}

	else if (surface_id == 1) {
		decay_coeff = 0.13;
	}
	else if (surface_id == 2) {
		decay_coeff = 0.19;
	}
	else if (surface_id == 3) {
		decay_coeff = 0.14;
	}
	else if (surface_id == 4) {
		decay_coeff = 0.18;
	}
	else
		decay_coeff = 0.16;

	left_energy = (1 - decay_coeff) * energy;

	return left_energy;
}


// you can not make this function 'global'-> it's not allowed
// but you can do recursion in device function
__device__ void processRay(Ray* dev_ray, Room* d_room) {

	// DEBUG INFO: point is a pointer to dev_ray->ray_start_point
	/*Process Here: 
	a) get start point of the ray 
	b) copy the start point to old point
	c) check if this segment of ray has intersection with sphere or not
	d) if there is no intersection between sphere and the segment, then cal reflection and intersection point
		IMPORTANT NOTE HERE: You call the setRayStartPoint function inside calValidSurfaceID function, which assign
							 intersection point to dev_ray->ray_start_point, and since point var in processRay function is the pointer
							 of dev_ray->ray_start_point (that's why you copy the data inside the point to the other var old point)*/
	while (fabs(getEnergy(dev_ray)) > 0.01) // || isPointOnEdge(d_room, dev_ray->ray_start_point)
	{
		float* point = getStartingPoint(dev_ray);
		for (int i = 0; i < 3; i++)
		{
			printf("this is the start point: %f\n", point[i]);

		}
		// if you do not copy point data to old_point, after you call function setStartingPoint(intersection_point),
		// you are actually set intersection_point data to point too. So under this you got intersection_point == point
		float old_point[3];
		for (int i = 0; i < 3; i++)
		{
			old_point[i] = point[i];
			printf("this is the OLD POINT: %f\n", old_point[i]);
		}
		float energy = getEnergy(dev_ray);
		printf("this is the start energy: %f\n", energy);

		/*if (fabs(energy) < 0.05 || isPointOnEdge(d_room, dev_ray->ray_start_point)) {
			setEnergy(dev_ray, 0);
			setDistance(dev_ray, 0);
			return;
		}*/

		if (isInSphere(dev_ray, d_room)) {
			float* listen_point = getListPos(d_room);
			// here cal the distance between listen point and source
			float distance;
			distance = calPointsDistance(dev_ray->ray_start_point, listen_point);
			float total_dist = distance + getDistance(dev_ray);
			setDistance(dev_ray, total_dist);
			return;

		}

		else {
			float* incident_ray_direction = getRayDirection(dev_ray);

			for (int i = 0; i < 3; i++)
			{
				printf("this is the incident: %f\n", incident_ray_direction[i]);

			}
			int ray_sign = getSign(dev_ray);
			printf("this sign: %d\n", ray_sign);
			int surface_id = getCurrentValudSurfaceID(dev_ray, dev_ray->ray_start_point, d_room);
			printf("this VALID: %d\n", surface_id);
			if (surface_id == -1)
			{
				printf("no valid surface");
			}

			float* reflected_ray_direction = calReflecRay(incident_ray_direction, surface_id);

			for (int i = 0; i < 3; i++)
			{
				printf("this is the reflec: %f\n", reflected_ray_direction[i]);

			}
			float* intersection_point = getStartingPoint(dev_ray);
			for (int i = 0; i < 3; i++)
			{

				printf("this is the intersection: %f\n", intersection_point[i]);
			}
			float dist_one_reflection = calPointsDistance(old_point, intersection_point);
			printf("this is the dist between one reflec: %f\n", dist_one_reflection);
			float traveled_dist = dist_one_reflection + getDistance(dev_ray);
			printf("travaled_dist is : %f\n", traveled_dist);
			float energy = absorbEnergy(getEnergy(dev_ray), surface_id);
			float left_energy = ray_sign * energy;
			printf("this is the left_energy: %f\n", left_energy);

			setStartingPoint(dev_ray, intersection_point);
			setEnergy(dev_ray, left_energy);
			setDistance(dev_ray, traveled_dist);
			setSign(dev_ray, -ray_sign);
			setRayDirection(dev_ray, reflected_ray_direction);

			//processRay(dev_ray, d_room);
		}
	}
	//float* point = getStartingPoint(dev_ray);
	//for (int i = 0; i < 3; i++)
	//{
	//	printf("this is the start point: %f\n", point[i]);

	//}
	//// if you do not copy point data to old_point, after you call function setStartingPoint(intersection_point),
	//// you are actually set intersection_point data to point too. So under this you got intersection_point == point
	//float old_point[3];
	//for (int i = 0; i < 3; i++)
	//{
	//	old_point[i] = point[i];
	//	printf("this is the OLD POINT: %f\n", old_point[i]);
	//}
	//float energy = getEnergy(dev_ray);
	//printf("this is the start energy: %f\n", energy);

	//if (fabs(energy) < 0.05 || isPointOnEdge(d_room,dev_ray->ray_start_point)) {
	//	setEnergy(dev_ray, 0);
	//	setDistance(dev_ray, 0);
	//	return;
	//}

	//if (isInSphere(dev_ray, d_room)) {
	//	float* listen_point = getListPos(d_room);
	//	// here cal the distance between listen point and source
	//	float distance;
	//	distance = calPointsDistance(dev_ray->ray_start_point, listen_point);
	//	float total_dist = distance + getDistance(dev_ray);
	//	setDistance(dev_ray,total_dist);
	//	return;

	//}

	//else {
	//	float* incident_ray_direction = getRayDirection(dev_ray);
	//	
	//	for (int i = 0; i < 3; i++)
	//	{
	//		printf("this is the incident: %f\n", incident_ray_direction[i]);
	//		
	//	}
	//	int ray_sign = getSign(dev_ray);
	//	printf("this sign: %d\n", ray_sign);
	//	int surface_id = getCurrentValudSurfaceID(dev_ray, dev_ray->ray_start_point, d_room);
	//	printf("this VALID: %d\n", surface_id);
	//	if (surface_id == -1)
	//	{
	//		printf("no valid surface");
	//	}
	//	
	//	float* reflected_ray_direction = calReflecRay(incident_ray_direction, surface_id);
	//	
	//	for (int i = 0; i < 3; i++)
	//	{
	//		printf("this is the reflec: %f\n", reflected_ray_direction[i]);
	//		
	//	}
	//	float* intersection_point = getStartingPoint(dev_ray);
	//	for (int i = 0; i < 3; i++)
	//	{
	//		
	//		printf("this is the intersection: %f\n", intersection_point[i]);
	//	}
	//	float dist_one_reflection = calPointsDistance(old_point, intersection_point);
	//	printf("this is the dist between one reflec: %f\n", dist_one_reflection);
	//	float traveled_dist = dist_one_reflection + getDistance(dev_ray);
	//	printf("travaled_dist is : %f\n", traveled_dist);
	//	float energy = absorbEnergy(getEnergy(dev_ray), surface_id);
	//	float left_energy = ray_sign * energy;
	//	printf("this is the left_energy: %f\n", left_energy);

	//	setStartingPoint(dev_ray, intersection_point);
	//	setEnergy(dev_ray, left_energy);
	//	setDistance(dev_ray, traveled_dist);
	//	setSign(dev_ray, -ray_sign);
	//	setRayDirection(dev_ray, reflected_ray_direction);

	//	processRay(dev_ray, d_room);
	//}

	
}

__global__ void processAllRays(Ray** dev_ray, Room* d_room) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	printf("this is the idx: %d\n", idx);

	processRay(dev_ray[idx], d_room);
}
// these functions are designed for testing cuda
__global__ void print(Ray** dev_ray, Room*d_room)

{
	
	// test for calDistance between two points function
	/*float a[3] = { 1.2, 5, 7 };
	float b[3] = { 2.4, 5.5, -1.3 };
	float dist = calPointsDistance(a, b);
	printf("this is the distance between to point: %f\n", dist);*/

	// test for isInSphere
	//bool ok = isInSphere(dev_ray, d_room);

	// test for calReflecRay
	/*float* reflected_ray;
	reflected_ray = calReflecRay(dev_ray->ray_direction, 4);*/

	// test for getValidSurface
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	//getCurrentValudSurfaceID(dev_ray, getStartingPoint(dev_ray), d_room);
	printf("this is the idx: %d\n", idx);

	//printf("this is the ray_energy: %d\n",(*(dev_ray[idx])).ray_sign);
	processRay(dev_ray[idx], d_room);
	//for (int i = 0; i < 3; i++)
	//{

	//	//printf("THIS IS RAY_DIREC: %f", dev_ray->ray_direction[i]);

	//	printf("ray_direction: %f\n", dev_ray[idx]->ray_direction[i]);

	//	//printf("reflected_Ray: %f\n", reflected_ray[i]);
	//}

}

int main(void)
	{
		hipError_t err_ray = hipSuccess;
		hipError_t err_room = hipSuccess;
		hipError_t err_all_rays = hipSuccess;
		
		int num_rays = 5;
		float** all_rays_directions;
		all_rays_directions = generateRays(num_rays);

		// define the sound source position, listener position, adn room dimension info
		float source[3] = { 1.2, 3.6, 5.9 };
		float l_position[3] = { 5.3, 7.5, 8.2 };
		float room_dim[3] = { 9.5, 7.3, 11.4 };
	
		// allocate host ptr for room and ray in CPU
		Room* h_room_ptr = new Room(source, l_position, room_dim);
		Ray** h_all_rays_ptr = new Ray*[num_rays];


		if (h_room_ptr == NULL)
		{
			fprintf(stderr, "Failed to allocate host vectors!\n");
			exit(EXIT_FAILURE);
		}
		if (h_all_rays_ptr == NULL)
		{
			fprintf(stderr, "Failed to allocate host all_rays!\n");
			exit(EXIT_FAILURE);
		}

		// in order to create device array of pointers to device array, first

		// create host array of pointers to device arrays, and after that copy it

		// into device array of pointers to device arrays.
		for (int i = 0; i < num_rays; i++)
		{   

			Ray* h_ray = new Ray(source, all_rays_directions[i]);  // h_ray is the host array intermediate one
			hipMalloc((void**)&h_all_rays_ptr[i], sizeof(Ray));
			hipMemcpy(h_all_rays_ptr[i], h_ray, sizeof(Ray), hipMemcpyHostToDevice); // copy it to h_all (device array of pointers)

		}

		
		size_t size_one_ray = sizeof(Ray);
		size_t roomsize = sizeof(Room);
		Ray** d_all_rays = NULL;
		Room* d_room_ptr = NULL;

		err_all_rays = hipMalloc(&d_all_rays, sizeof(Ray*)*num_rays);
		err_room = hipMalloc((void**)&d_room_ptr, roomsize);
	
		if (err_room != hipSuccess)
		{
			fprintf(stderr, "Failed to allocate device memory for room(error code %s)!\n", hipGetErrorString(err_room));
			exit(EXIT_FAILURE);
		}
		if (err_all_rays != hipSuccess)
		{
			fprintf(stderr, "Failed to allocate device memory for all rays(error code %s)!\n", hipGetErrorString(err_all_rays));
			exit(EXIT_FAILURE);
		}

		// Copy the host input ray and room in host memory to the device input ray and room in
		// device memory

		printf("Copy input data from the host memory to the CUDA device\n");
		err_all_rays = hipMemcpy(d_all_rays, h_all_rays_ptr, num_rays*sizeof(Ray*), hipMemcpyHostToDevice); // both h_ and d_ are device arrays of pointers
		err_room = hipMemcpy(d_room_ptr, h_room_ptr, roomsize, hipMemcpyHostToDevice);
		

		if (err_room != hipSuccess)
		{
			fprintf(stderr, "Failed to room from host to device (error code %s)!\n", hipGetErrorString(err_room));
			exit(EXIT_FAILURE);
		}

		
		// Launch the processAllRays function CUDA Kernel
		processAllRays << <1, num_rays>> >( d_all_rays, d_room_ptr);
		err_all_rays = hipGetLastError();
		
		if (err_all_rays != hipSuccess)
		{
			fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err_all_rays));
			exit(EXIT_FAILURE);
		}

		// Copy the ray device in device memory to the host result ray
		// in host memory. Here do not need to copy room data back to CPU
		printf("Copy output data from the CUDA device to the host memory\n");
		// TODO: copy data in device back to host.
		//err_all_rays = hipMemcpy(h_all_rays_ptr, d_all_rays, size, hipMemcpyDeviceToHost);
		
		/*if (err_all_rays != hipSuccess)
		{
			fprintf(stderr, "Failed to copy all ray info from device to host (error code %s)!\n", hipGetErrorString(err_all_rays));
			exit(EXIT_FAILURE);
		}
*/

		// Free device global memory
		//err_ray = hipFree(d_ray);
		err_room = hipFree(d_room_ptr);
		err_all_rays = hipFree(d_all_rays);
		hipFree(h_all_rays_ptr);

		if (err_room != hipSuccess)
		{
			fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err_room));
			exit(EXIT_FAILURE);
		}
		if (err_all_rays != hipSuccess)
		{
			fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err_all_rays));
			exit(EXIT_FAILURE);
		}


		// Free host memory for ray and room
		
		free(h_room_ptr);
		//free(h_all_rays_ptr);

		//err_ray = hipDeviceReset();
		err_room = hipDeviceReset();
		err_all_rays = hipDeviceReset();
		/*if (err_ray != hipSuccess)
		{
			fprintf(stderr, "Failed to deinitialize the device for ray! error=%s\n", hipGetErrorString(err_ray));
			exit(EXIT_FAILURE);
		}*/
		if (err_room != hipSuccess)
		{
			fprintf(stderr, "Failed to deinitialize the device for room! error=%s\n", hipGetErrorString(err_room));
			exit(EXIT_FAILURE);
		}
		if (err_all_rays != hipSuccess)
		{
		fprintf(stderr, "Failed to deinitialize the device for ray! error=%s\n", hipGetErrorString(err_all_rays));
		exit(EXIT_FAILURE);
		}

		printf("Done\n");
		return 0;
}; 
